
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {  // The qualifier __global__ tells the compiler
                                      // that the function will be called from the CPU and executed on the GPU.
  printf("Hello World from GPU from thread %d\n", threadIdx.x);
}

int main(void) {
  printf("Hello World from CPU!\n");

  // Triple angle brackets mark a call from the host thread to the code on the device side. A kernel is
  // executed by an array of threads and all threads run the same code. The parameters within the triple
  // angle brackets are the execution configuration, which specifies how many threads will execute the
  // kernel. In this example, you will run 10 GPU threads.
  helloFromGPU <<<1, 10>>>();
  hipDeviceReset();   // The function cudaDeviceReset() will explicitly destroy and
                       // clean up all resources associated with the current device in the current process.
  return 0;
}
